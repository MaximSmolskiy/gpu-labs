#include "hip/hip_runtime.h"
#include <chrono>
#include <cinttypes>
#include <iostream>
#include <random>

#include "hip/hip_runtime.h"
#include ""

size_t const BLOCK_SIZE = 32;

float *randomMatrix(float const minimum_value, float const maximum_value, size_t const n) {
    std::random_device random_device;
    std::mt19937 generator(random_device());
    std::uniform_real_distribution<float> const uniform_real_distribution(minimum_value, maximum_value);
    float *const random_matrix = new float[n * n];
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            random_matrix[i * n + j] = uniform_real_distribution(generator);
        }
    }
    return random_matrix;
}

__global__ void gpuSharedMemoryMatrixMultiplicationKernel(float const *const a, float const *const b, float *const c, size_t const n) {
    size_t const i = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    size_t const j = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float c_i_j = 0;
    for (size_t submatrix_index = 0; submatrix_index * BLOCK_SIZE < n; ++submatrix_index) {
        __shared__ float submatrix_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float submatrix_b[BLOCK_SIZE][BLOCK_SIZE];

        submatrix_a[threadIdx.y][threadIdx.x] = 0;
        submatrix_b[threadIdx.y][threadIdx.x] = 0;

        size_t const submatrix_a_j = submatrix_index * BLOCK_SIZE + threadIdx.x;
        if (i < n && submatrix_a_j < n) {
            submatrix_a[threadIdx.y][threadIdx.x] = a[i * n + submatrix_a_j];
        }

        size_t const submatrix_b_i = submatrix_index * BLOCK_SIZE + threadIdx.y;
        if (submatrix_b_i < n && j < n) {
            submatrix_b[threadIdx.y][threadIdx.x] = b[submatrix_b_i * n + j];
        }

        __syncthreads();

        for (size_t k = 0; k < BLOCK_SIZE; ++k) {
            c_i_j += submatrix_a[threadIdx.y][k] * submatrix_b[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (i < n && j < n) {
        c[i * n + j] = c_i_j;
    }
}

__global__ void gpuWarpIntrinsicsMatrixMultiplicationKernel(float const *const a, float const *const b, float *const c, size_t const n) {
    size_t const start_i = blockIdx.y * BLOCK_SIZE;
    size_t const j = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float column_c[BLOCK_SIZE] = { 0 };
    for (size_t submatrix_index = 0; submatrix_index * BLOCK_SIZE < n; ++submatrix_index) {
        for (size_t k = 0; k < BLOCK_SIZE; ++k) {
            float a_i_k = 0;
            float b_k_j = 0;

            size_t const i = start_i + threadIdx.x;
            size_t const submatrix_a_j = submatrix_index * BLOCK_SIZE + k;
            if (i < n && submatrix_a_j < n) {
                a_i_k = a[i * n + submatrix_a_j];
            }

            size_t const submatrix_b_i = submatrix_index * BLOCK_SIZE + k;
            if (submatrix_b_i < n && j < n) {
                b_k_j = b[submatrix_b_i * n + j];
            }

            for (size_t l = 0; l < BLOCK_SIZE; ++l) {
                column_c[l] += __shfl_sync(0xFFFFFFFF, a_i_k, l) * b_k_j;
            }
        }
    }

    for (size_t l = 0; l < BLOCK_SIZE; ++l) {
        size_t const i = start_i + l;
        if (i < n && j < n) {
            c[i * n + j] = column_c[l];
        }
    }
}

void gpuMultiplyMatrices(float const *const a, float const *const b, float *const c, size_t const n, bool const warp_intrinsics) {
    float *device_a;
    float *device_b;
    float *device_c;

    hipMalloc(&device_a, n * n * sizeof(float));
    hipMalloc(&device_b, n * n * sizeof(float));
    hipMalloc(&device_c, n * n * sizeof(float));

    hipMemcpy(device_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);

    auto const start = std::chrono::high_resolution_clock::now();

    if (warp_intrinsics) {
        dim3 const block_dimensions(BLOCK_SIZE, 1);
        dim3 const grid_dimensions((n + block_dimensions.x - 1) / block_dimensions.x, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);
        gpuWarpIntrinsicsMatrixMultiplicationKernel<<<grid_dimensions, block_dimensions>>>(device_a, device_b, device_c, n);
    } else {
        dim3 const block_dimensions(BLOCK_SIZE, BLOCK_SIZE);
        dim3 const grid_dimensions((n + block_dimensions.x - 1) / block_dimensions.x, (n + block_dimensions.y - 1) / block_dimensions.y);
        gpuSharedMemoryMatrixMultiplicationKernel<<<grid_dimensions, block_dimensions>>>(device_a, device_b, device_c, n);
    }
    hipDeviceSynchronize();

    auto const end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> const elapsed_seconds = end - start;

    if (warp_intrinsics) {
        std::cout << "GPU warp intrinsics elapsed time = ";
    } else {
        std::cout << "GPU shared memory elapsed time = ";
    }
    std::cout << elapsed_seconds.count() << " seconds" << std::endl;

    hipMemcpy(c, device_c, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
}

float maximumMatrixDeviation(float const *const a, float const *const b, size_t const n) {
    float maximum_matrix_deviation = 0;
    for (size_t i = 0; i < n; ++i) {
        for (size_t j = 0; j < n; ++j) {
            maximum_matrix_deviation = std::max(maximum_matrix_deviation, std::abs(a[i * n + j] - b[i * n + j]));
        }
    }
    return maximum_matrix_deviation;
}

int main(int argc, char *argv []) {
    size_t const n = std::strtoumax(argv[1], nullptr, 10);
    std::cout << "n = " << n << std::endl;

    float const MINIMUM_VALUE = -10;
    float const MAXIMUM_VALUE = 10;
    float const *const a = randomMatrix(MINIMUM_VALUE, MAXIMUM_VALUE, n);
    float const *const b = randomMatrix(MINIMUM_VALUE, MAXIMUM_VALUE, n);

    float *const gpu_shared_memory_c = new float[n * n];
    gpuMultiplyMatrices(a, b, gpu_shared_memory_c, n, false);

    float *const gpu_warp_intrinsics_c = new float[n * n];
    gpuMultiplyMatrices(a, b, gpu_warp_intrinsics_c, n, true);

    delete[] a;
    delete[] b;

    std::cout << "maximum GPU shared memory and warp intrinsics matrix deviation = " << maximumMatrixDeviation(gpu_shared_memory_c, gpu_warp_intrinsics_c, n) << std::endl;

    delete[] gpu_shared_memory_c;
    delete[] gpu_warp_intrinsics_c;
}
